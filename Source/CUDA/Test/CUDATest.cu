/// \file CUDATest.cu
/// \author Ольга Рыжкова
#include <RJF/Default.hpp>
#include <CppUTest/TestHarness.h>
#include <RJF/TestFunctions.hpp>
#include <RJF/CUDA.hpp>

#include <cmath>
#include <fstream>
#include <iostream>
#include <string>
#include <vector>
#include <chrono>

TEST_GROUP(RJFTestGroup)
{ 

};

TEST(RJFTestGroup, testcuda)
{ 
    int m = 1024;
    int n = 100;

    Array X;
    RRTC::Ambiguity::ReadArray("byteArrayX.bin", X);

    Array Y;
    RRTC::Ambiguity::ReadArray("byteArrayY.bin", Y);

    Array expd;
    RRTC::Ambiguity::ReadArray("byteArrayY.bin", expd);
 
    int size = n * m * sizeof(RRTC::Complex);
     
    RRTC::Complex * dX = nullptr;
    RRTC::Complex * dY = nullptr;
     
    hipMalloc(&dX, size);
    hipMalloc(&dY, size);
     
    hipMemcpy(dX, X.data(), size, hipMemcpyHostToDevice);

    RRTC::Ambiguity::CudaRJF filter(0.9);
    
    auto begin = std::chrono::steady_clock::now();
    filter.Execute(dY, dX, m, n);
    auto end = std::chrono::steady_clock::now();

       auto elapsed_ms = std::chrono::duration_cast<std::chrono::microseconds>(end - begin);
       std::cout << "The time: " << elapsed_ms.count() << " ";

    hipMemcpy(Y.data(), dY, size, hipMemcpyDeviceToHost);

    float STDREIM = RRTC::Ambiguity::calculatingDeviation(m, n, Y.data(), expd.data());

    double checking = 1E-5;

    DOUBLES_EQUAL(0, STDREIM, checking); 

    hipFree(dX);
    hipFree(dY);
}
