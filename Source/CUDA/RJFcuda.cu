#include "hip/hip_runtime.h"
/// \file RJFcuda.cu
/// \author Ольга Рыжкова
#include <RJF/Default.hpp>
#include <RJF/Interface.hpp>
#include <RJF/Data.hpp>
#include <RJF/CUDA.hpp>


namespace Device
{
    using Complex = RRTC::Complex;

     __device__ Complex Plus(const Complex& left, const Complex& right)
     {
          return {left.re + right.re, left.im + right.im};
     } 

     __device__ Complex Minus(const Complex& left, const Complex& right)
     {
          return {left.re - right.re, left.im - right.im};
     } 

     __device__ Complex Mult(const Complex& left, const float k2)
     {
          return {left.re * k2, left.im * k2};
     } 


    __global__ void RJF(Complex* Y, const Complex* X, std::size_t M, std::size_t N, float beta)
    {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx < M)
        {
            const RRTC::Complex* x = (X + idx * N);
            RRTC::Complex* y = (Y + idx * N);
            
            y[0] = {0, 0};
            
            for (std::size_t n = 1; n < N; ++n)
            {         
                 y[n] = Plus(Mult(y[n - 1], beta), Minus(x[n], x[n - 1]));
            }   
        }
    }    
} //Device

namespace RRTC
{
    namespace Ambiguity
    {
        void CudaRJF::Execute(Complex* Y, const Complex* X, std::size_t M, std::size_t N)
        {
            Device::RJF<<<std::ceil((double(M) * N)/128), 128>>>(Y, X, M, N, this-> beta);
        }
    } //Ambiguity
} //RRTC







